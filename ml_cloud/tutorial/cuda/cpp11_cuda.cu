#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

#include "range.hpp"

using namespace util::lang;

template <typename T>
using step_range = typename range_proxy<T>::step_range_proxy;


template <typename T>
__device__ step_range<T> grid_stride_range(T begin, T end) {
    begin += blockDim.x * blockIdx.x + threadIdx.x;
    return range(begin, end).step(gridDim.x * blockDim.x);
}

template <typename T, typename Predicate>
__device__ void count_if (int *count, T *data, int n, Predicate p) {
    for (auto i:grid_stride_range(0, n)) {
        if (p(data[i])) atomicAdd(count, 1);
    }
}

__global__ void xyzw_frequency(int *count, char *text, int n) {
    const char letters[] {'x','y','z', 'w'};

    count_if(count ,text, n, [&](char c) {
        for (const auto x : letters) {
            if (c==x) return true;
        }
        return false;
    });
}

__global__ void xyzw_frequency_thrust_device(int *count, char *text, int n) {
    const char letters[] {'x', 'y', 'z', 'w'};
    *count = thrust::count_if(thrust::device, text, text+n, [=](char c) {
        for (const auto x: letters) {
            if (c == x) {
                return true;
            }
        }
        return false;
    });
}


int main(int argc, char** argv) {
    const char* filename = sdkFindFilePath("warandpeace.txt", argv[0]);

    int numBytes = 16* 1048576;
    char *h_text = (char*) malloc(numBytes);

    int devID = findCudaDevice(argc, (const char**)argv);
    
    char *d_text;
    checkCudaErrors(hipMalloc((void**)&d_text, numBytes));

    FILE *fp = fopen(filename, "r");
    if (fp == NULL) {
        printf("Cannot find the input text file\n. Exiting..\n");
        return EXIT_FAILURE;
    }
    int len = (int)fread(h_text, sizeof(char), numBytes, fp);
    fclose(fp);
    std::cout << "Read " << len << " byte corpus from " << filename << std::endl;

    checkCudaErrors(hipMemcpy(d_text, h_text, len, hipMemcpyHostToDevice));

    int count = 0;
    int *d_count;
    checkCudaErrors(hipMalloc(&d_count, sizeof(int)));
    checkCudaErrors(hipMemset(d_count, 0, sizeof(int)));

    xyzw_frequency<<<8, 256>>>(d_count, d_text, len);
    xyzw_frequency_thrust_device<<<1,1>>>(d_count, d_text, len);
    checkCudaErrors(
        hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    
    std::count << "counted " <<count << "instances of 'x', 'y', 'z', 'w' in \" " << filename << "\"" << std::endl;

    checkCudaErrors(hipFree(d_count));
    checkCudaErrors(hipFree(d_text));

    return EXIT_SUCCESS;
}